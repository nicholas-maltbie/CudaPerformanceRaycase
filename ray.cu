#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "hip/hip_runtime.h"
#include "common/book.h"
#include "common/cpu_bitmap.h"

#define MAX_SPHERES 1000

#define rnd( x ) (x * rand() / RAND_MAX)
#define INF     2e10f

struct Sphere {
    float   r,b,g;
    float   radius;
    float   x,y,z;
    __device__ float hit( float ox, float oy, float *n ) {
        float dx = ox - x;
        float dy = oy - y;
        if (dx*dx + dy*dy < radius*radius) {
            float dz = sqrtf( radius*radius - dx*dx - dy*dy );
            *n = dz / sqrtf( radius * radius );
            return dz + z;
        }
        return -INF;
    }
};

__constant__ Sphere sconstant[MAX_SPHERES];

__global__ void kernel( Sphere *sglobal, unsigned char *ptr, int dim,
    int spheres, bool isGlobal) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
    float   ox = (x - dim/2);
    float   oy = (y - dim/2);

    Sphere* s = sconstant;
    if (isGlobal) {
        s = sglobal;
    }

    float   r=0, g=0, b=0;
    float   maxz = -INF;
    for(int i=0; i<spheres; i++) {
        float   n;
        float   t = s[i].hit( ox, oy, &n );
        if (t > maxz) {
            float fscale = n;
            r = s[i].r * fscale;
            g = s[i].g * fscale;
            b = s[i].b * fscale;
            maxz = t;
        }
    } 

    ptr[offset*4 + 0] = (int)(r * 255);
    ptr[offset*4 + 1] = (int)(g * 255);
    ptr[offset*4 + 2] = (int)(b * 255);
    ptr[offset*4 + 3] = 255;
}

// globals needed by the update routine
struct DataBlock {
    unsigned char   *dev_bitmap;
};

int main( int argc, char** argv ) {
    if (argc != 6) {
        printf("Usage: %s DIM SPHERES NUM-THREADS RENDER GLOBAL\n", argv[0]);
        printf("  DIM - Dimension of image (square) in pixels\n");
        printf("  SPHERES - Number of spheres to draw\n");
        printf("  NUM-THREADS - Number of threads to use per block\n");
        printf("  RENDER - Value of 0 for do not render, 1 for render\n");
        printf("  GLOBAL - Value of 0 for constant memory, 1 for global\n");
        return 1;
    }

    int dim = atoi(argv[1]);
    int spheres = atoi(argv[2]);
    int num_threads = atoi(argv[3]);
    bool render = atoi(argv[4]) == 1;
    bool isGlobal = atoi(argv[5]) == 1;

    DataBlock   data;
    // capture the start time
    hipEvent_t     start, stop;
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );
    HANDLE_ERROR( hipEventRecord( start, 0 ) );

    CPUBitmap bitmap( dim, dim, &data );
    unsigned char   *dev_bitmap;

    // allocate memory on the GPU for the output bitmap
    HANDLE_ERROR( hipMalloc( (void**)&dev_bitmap,
                              bitmap.image_size() ) );

    // allocate temp memory, initialize it, copy to constant
    // memory on the GPU, then free our temp memory
    Sphere *temp_s = (Sphere*)malloc( sizeof(Sphere) * spheres );
    for (int i=0; i<spheres; i++) {
        temp_s[i].r = rnd( 1.0f );
        temp_s[i].g = rnd( 1.0f );
        temp_s[i].b = rnd( 1.0f );
        temp_s[i].x = rnd( 1000.0f ) - 500;
        temp_s[i].y = rnd( 1000.0f ) - 500;
        temp_s[i].z = rnd( 1000.0f ) - 500;
        temp_s[i].radius = rnd( 100.0f ) + 20;
    }

    Sphere          *sGlobal;
    if (isGlobal) {
        HANDLE_ERROR( hipMalloc( (void**)&sGlobal,
                                  sizeof(Sphere) * spheres ) );
        HANDLE_ERROR( hipMemcpy( sGlobal, temp_s,
                                    sizeof(Sphere) * spheres,
                                    hipMemcpyHostToDevice ) );
    }
    else {
        HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( sconstant), temp_s, 
                                    sizeof(Sphere) * spheres) );
    }
    free( temp_s );

    // generate a bitmap from our sphere data
    dim3    grids(dim/num_threads,dim/num_threads);
    dim3    threads(num_threads,num_threads);
    kernel<<<grids,threads>>>( sGlobal, dev_bitmap, dim, spheres, isGlobal);

    // copy our bitmap back from the GPU for display
    HANDLE_ERROR( hipMemcpy( bitmap.get_ptr(), dev_bitmap,
                              bitmap.image_size(),
                              hipMemcpyDeviceToHost ) );

    // get stop time, and display the timing results
    HANDLE_ERROR( hipEventRecord( stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( stop ) );
    float   elapsedTime;
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );
    //printf( "Time to generate:  %3.1f ms\n", elapsedTime );
    printf( "%3.1f\n", elapsedTime );

    HANDLE_ERROR( hipEventDestroy( start ) );
    HANDLE_ERROR( hipEventDestroy( stop ) );

    HANDLE_ERROR( hipFree( dev_bitmap ) );

    // display
    if (render) {
        bitmap.display_and_exit();
    }
}

